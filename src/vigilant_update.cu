#include "hip/hip_runtime.h"
#include "vigilant_update.h"

#define THREADS_PER_BLOCK 1024
#define FINITE_DIFF_CONST 0.145f


#ifdef CUDA_EMULATE
static float rsqrtf(float value) {
    return 1.0f / sqrt(value);
}
#endif


TOPLEVEL_SIMP static void statsUpdateKernel(statsUpdateArgs args) {

    int idx = BLOCK_IDX.x * BLOCK_DIM.x + THREAD_IDX.x;
    if (idx >= args.numElems) {
        return;
    }

    float grad = args.grads[idx];

    // Update the statistics collector
    float sampleSize = args.sampleSize[idx];
    float mean = args.mean[idx];
    float meanSq = args.meanSq[idx];
    
    float sampleFrac = 1.0f / sampleSize;
    mean = (1.0f - sampleFrac) * mean + sampleFrac * grad;
    meanSq = (1.0f - sampleFrac) * meanSq + sampleFrac * (grad * grad);
    if (meanSq <= 0.0f) {
        meanSq = 1.0f;
    }

    float sqMean = mean * mean;
    float var = meanSq - sqMean;
    sampleSize += 2 * (sqMean * sampleSize < var) - 1;
    if (sampleSize < args.minSample) {
        sampleSize = args.minSample;
    }

    args.sampleSize[idx] = sampleSize;
    args.mean[idx] = mean;
    args.meanSq[idx] = meanSq;

    float rmsDenom = rsqrtf(meanSq);
    float weight = sqMean * rmsDenom;
    args.weight[idx] = weight;
    args.weightedSampleSize[idx] = weight * sampleSize;

    // Update the time 
    int time = args.time[idx];
    float acceler = 0.0f;

    time += 1;
    if (time > sampleSize / 2) {
        float oldMean = args.oldMean[idx];
        float oldMeanSign = (oldMean > 0.0f) ? 1.0f : -1.0f;
        acceler = 2 * (oldMeanSign * (oldMean - mean) <
                       oldMeanSign * (oldMean * FINITE_DIFF_CONST)) - 1;

        time = 0;
        args.oldMean[idx] = mean;
    }
    args.time[idx] = time;

    args.weightedAcceler[idx] = weight * acceler;
}

void statsUpdateImpl(statsUpdateArgs args) {

    int threadsPerBlock = THREADS_PER_BLOCK;
    #ifdef CUDA_EMULATE
        threadsPerBlock = args.numElems;
    #endif

    LAUNCH_TOPLEVEL_SIMP(
        statsUpdateKernel,
        LAUNCH_PARAMS(
            TO_DIM3((args.numElems + threadsPerBlock - 1) / threadsPerBlock),
            TO_DIM3(threadsPerBlock)
        ),
        args
    );
}


TOPLEVEL_SIMP static void stepUpdateKernel(stepUpdateArgs args) {

    int idx = BLOCK_IDX.x * BLOCK_DIM.x + THREAD_IDX.x;
    if (idx >= args.numElems) {
        return;
    }

    float grad = args.grads[idx];
    float stepDecay = args.stepDecay;
    float stepFactorOverSampleSize = args.stepFactorOverSampleSize;

    // Compute a new step size
    float step = args.step[idx];
    float meanSq = args.meanSq[idx];
    float rsqrtMeanSq = rsqrtf(meanSq);

    step = stepDecay * step + (1.0 - stepDecay) * grad * rsqrtMeanSq;

    // Update the data
    float update = stepFactorOverSampleSize * step;

    float deviation = args.deviation[idx];
    float exploreUpdate = args.baseLr * grad * rsqrtMeanSq;
    float newDeviation = args.deviationDecay * deviation - exploreUpdate;

    args.step[idx] = step;
    args.data[idx] += (-update - deviation + newDeviation);
    args.deviation[idx] = newDeviation;
}


void stepUpdateImpl(stepUpdateArgs args) {
    int threadsPerBlock = THREADS_PER_BLOCK;
    #ifdef CUDA_EMULATE
        threadsPerBlock = args.numElems;
    #endif

    LAUNCH_TOPLEVEL_SIMP(
        stepUpdateKernel,
        LAUNCH_PARAMS(
            TO_DIM3((args.numElems + threadsPerBlock - 1) / threadsPerBlock),
            TO_DIM3(threadsPerBlock)
        ),
        args
    );
}
